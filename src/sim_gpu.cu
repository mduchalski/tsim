#include "hip/hip_runtime.h"
#include "sim_gpu.h"

#include <stdio.h>
#include <float.h>

#define MAX_PREDECESSORS 4
#define DECELL_MAX 10.0

__device__ int find_pred(int *pred, const int node, const net_type net)
{
    int i = 0;
    for(int j = 0; j < net.nodes_n; j++)
        if(net.weights[j*net.nodes_n + node])
            pred[i++] = j;
    return i;
}

__device__ bool simple_inter(const double t, const int from, const int through,
    const double timeout, const double offset, const net_type net)
{
    int pred[MAX_PREDECESSORS];
    int pred_n = find_pred(pred, through, net);
    return pred[(int)(fmod(t+offset, (double)pred_n*timeout) / timeout)] == from;
}

__device__ bool inter_open(const double t, const int from, const int through,
    const int to, const net_type net) 
{
    if(net.inters[through].type_id == SIMPLE)
        return simple_inter(t, from, through, 
            net.inters_params[net.inters[through].params_start],           // offset
            net.inters_params[net.inters[through].params_start + 1], net); // timeout
    
    return true; // ALWAYS_OPEN and invalid entries   
}

__device__ bool agents_edge_cmp_gpu(const agent_state_type a, const agent_state_type b)
{
    if(a.prev != b.prev)
        return a.prev < b.prev;
    else if(a.next != b.next)
        return a.next < b.next;
    else return 0;
}

__device__ int first_on_next_edge(const int i, const agent_state_type *states, 
    const int *routes, const int ags_n) {
    agent_state_type t;
    t.prev = states[i].next;
    t.next = routes[states[i].route_pos];
    int l = 0, r = ags_n, m;
    while (l < r) {
        m = (l+r)/2;
        if(agents_edge_cmp_gpu(states[m], t))
            l = m+1;
        else r = m;
    }
    
    if(states[l].prev == t.prev && states[l].next == t.next)
        return l;
    return -1;
}

__device__ double idm_accel(const agent_state_type state, const agent_params_type params,
    const double x_ahead, const double v_ahead) {
    double ss = params.s0 + state.v*params.T + state.v*(state.v-v_ahead)/(2*sqrt(params.a*params.b));
    return params.a*(1 - pow(state.v/params.v0, 4) - pow(ss/(x_ahead-state.x), 2));
}

__global__ void agent_sim(agent_state_type *states, const double t,
    const double t_step, const net_type net, const agent_state_type *states_prev,
    const agent_params_type *params, const int *routes, const int ags_count)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i >= ags_count)
        return;
    
    if(states_prev[i].next < 0)
        return; // agent is inactive
    
    double x_ahead, v_ahead;
    
    if(i+1 < ags_count && states_prev[i+1].next == states_prev[i].next && 
        states_prev[i].prev == states_prev[i+1].prev) {
        // there is an agent ahead on the same edge
        x_ahead = states_prev[i+1].x;
        v_ahead = states_prev[i+1].v;
    }
    else if(states_prev[i].route_pos == params[states_prev[i].uid].route_end) {
        // agent is approaching its destination with no agents ahead
        x_ahead = DBL_MAX;
        v_ahead = 0.0;
    }
    else if(inter_open(t, states_prev[i].prev, states_prev[i].next, -1, net)) {
        // there is a open intersection ahead of an agent
        int j = first_on_next_edge(i, states_prev, routes, ags_count);
        if(j != -1) {
            // there is an agent on the next edge
            x_ahead = net.weights[states_prev[i].prev*net.nodes_n + states_prev[i].next]+states_prev[j].x;
            v_ahead = states_prev[j].v;
        }
        else {
            x_ahead = DBL_MAX;
            v_ahead = 0;
        }
    }
    else {
        // there is a closed intersection ahead of an agent
        x_ahead = net.weights[states_prev[i].prev*net.nodes_n + states_prev[i].next];
        v_ahead = 0;
    }

    states[i].x += t_step*states_prev[i].v;
    double accel = idm_accel(states_prev[i], params[states_prev[i].uid], x_ahead, v_ahead);
    if(accel < -DECELL_MAX)
        accel = -DECELL_MAX;
    states[i].v += t_step*accel;
    if(states[i].v < 0.0)
        states[i].v = 0.0;


    if(states[i].x > net.weights[states_prev[i].prev*net.nodes_n + states_prev[i].next]) {
        if(states_prev[i].route_pos == params[states_prev[i].uid].route_end) {
            states[i].next = -1;

        }
        else {
            states[i].x -= net.weights[states_prev[i].prev*net.nodes_n + states_prev[i].next];
            states[i].prev = states[i].next;
            states[i].next = routes[states[i].route_pos];
            states[i].route_pos++;
        }
    }
}

#define SQR(x) x*x

void copy_to_gpu(net_type *net, agents_type *ags,
    const net_type *h_net, const agents_type *h_ags) 
{
    hipMalloc(&net->weights, SQR(h_net->nodes_n) * sizeof(*net->weights));
    hipMalloc(&net->inters, h_net->nodes_n * sizeof(*net->inters));
    hipMalloc(&net->inters_params, h_net->inters_params_len * sizeof(*net->inters_params));
    hipMalloc(&ags->states, h_ags->count * sizeof(*ags->states));
    hipMalloc(&ags->params, h_ags->count * sizeof(*ags->params));
    hipMalloc(&ags->routes, h_ags->routes_len * sizeof(*ags->routes));

    hipMemcpy(net->weights, h_net->weights,
        SQR(h_net->nodes_n) * sizeof(*net->weights), hipMemcpyHostToDevice);
    hipMemcpy(net->inters, h_net->inters,
        h_net->nodes_n * sizeof(*net->inters), hipMemcpyHostToDevice);
    hipMemcpy(net->inters_params, h_net->inters_params,
        h_net->inters_params_len * sizeof(*net->inters_params), hipMemcpyHostToDevice);
    hipMemcpy(ags->states, h_ags->states,
        h_ags->count * sizeof(*ags->states), hipMemcpyHostToDevice);
    hipMemcpy(ags->params, h_ags->params,
        h_ags->count * sizeof(*ags->params), hipMemcpyHostToDevice);
    hipMemcpy(ags->routes, h_ags->routes,
        h_ags->routes_len * sizeof(*ags->routes), hipMemcpyHostToDevice);
    net->nodes_n = h_net->nodes_n;
}

void dealloc_gpu(net_type *net, agents_type *ags)
{
    hipFree(net->weights);
    hipFree(net->inters);
    hipFree(net->inters_params);
    hipFree(ags->states);
    hipFree(ags->params);
    hipFree(ags->routes);
}

#define PER_BLOCK 1024

void sim_gpu(const char* out_filename, const double t_step, const double t_final,
    const net_type *h_net, const agents_type *h_ags)
{
    net_type net;
    agents_type ags;
    copy_to_gpu(&net, &ags, h_net, h_ags);
    agent_state_type *states, *states_prev;
    agent_state_type *states_host = (agent_state_type*)malloc(h_ags->count * sizeof(*states_host));
    hipMalloc(&states, h_ags->count * sizeof(*states));
    hipMalloc(&states_prev, h_ags->count * sizeof(*states_prev));
    hipMemcpy(states, ags.states, h_ags->count * sizeof(*states), hipMemcpyDeviceToDevice);
    FILE *f = fopen(out_filename, "wb");

    int steps = (int)floor(t_final / t_step);
    fwrite(&t_step, sizeof(t_step), 1, f);
    fwrite(&steps, sizeof(steps), 1, f);
    fwrite(&h_ags->count, sizeof(h_ags->count), 1, f);
    for(int i = 0; i < steps; i++) {
        hipMemcpy(states_host, states, h_ags->count * sizeof(*states), hipMemcpyDeviceToHost);
        sort_agents(states_host, h_ags->count);
        fwrite(states_host, h_ags->count * sizeof(*states), 1, f);
        hipMemcpy(states, states_host, h_ags->count * sizeof(*states), hipMemcpyHostToDevice);
        hipMemcpy(states_prev, states, h_ags->count * sizeof(*states), hipMemcpyDeviceToDevice);
        agent_sim<<<1 + h_ags->count/PER_BLOCK, PER_BLOCK>>>(states, (double)i*t_step,
            t_step, net, states_prev, ags.params, ags.routes, h_ags->count);
        hipDeviceSynchronize();
    }

    fclose(f);
    f = NULL;
    free(states_host);
    states_host = NULL;
    dealloc_gpu(&net, &ags);
    hipFree(states);
    hipFree(states_prev);
}