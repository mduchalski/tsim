#include "sim_gpu.h"

#include <stdio.h>

void copy_to_gpu(net_type *net, agents_type *ags,
    const net_type *h_net, const agents_type *h_ags) 
{
    hipMalloc(&net->weights, h_net->nodes_n * sizeof(*net->weights));
    hipMalloc(&net->inters, h_net->nodes_n * sizeof(*net->inters));
    hipMalloc(&net->inters_params, h_net->inters_params_len * sizeof(*net->inters_params));
    hipMalloc(&ags->states, h_ags->count * sizeof(*ags->states));
    hipMalloc(&ags->params, h_ags->count * sizeof(*ags->params));
    hipMalloc(&ags->routes, h_ags->routes_len * sizeof(*ags->routes));

    hipMemcpy(net->weights, h_net->weights,
        h_net->nodes_n * sizeof(*net->weights), hipMemcpyHostToDevice);
    hipMemcpy(net->inters, h_net->inters,
        h_net->nodes_n * sizeof(*net->inters), hipMemcpyHostToDevice);
    hipMemcpy(net->inters_params, h_net->inters_params,
        h_net->inters_params_len * sizeof(*net->inters_params), hipMemcpyHostToDevice);
    hipMemcpy(ags->states, h_ags->states,
        h_ags->count * sizeof(*ags->states), hipMemcpyHostToDevice);
    hipMemcpy(ags->params, h_ags->params,
        h_ags->count * sizeof(*ags->params), hipMemcpyHostToDevice);
    hipMemcpy(ags->routes, h_ags->routes,
        h_ags->routes_len * sizeof(*ags->routes), hipMemcpyHostToDevice);
}

void dealloc_gpu(net_type *net, agents_type *ags)
{
    hipFree(net->weights);
    hipFree(net->inters);
    hipFree(net->inters_params);
    hipFree(ags->states);
    hipFree(ags->params);
    hipFree(ags->routes);
}

void sim_gpu(const char* out_filename, const double t_step, const double t_final,
    const net_type *h_net, const agents_type *h_ags)
{
    net_type net;
    agents_type ags;
    copy_to_gpu(&net, &ags, h_net, h_ags);
    agent_state_type *states, *states_prev;
    hipMalloc(&states, h_ags->count * sizeof(*states));
    hipMalloc(&states_prev, h_ags->count * sizeof(*states_prev));
    hipMemcpy(states, ags.states, h_ags->count * sizeof(*states), hipMemcpyDeviceToDevice);
    dealloc_gpu(&net, &ags);
    hipFree(states);
    hipFree(states_prev);
}